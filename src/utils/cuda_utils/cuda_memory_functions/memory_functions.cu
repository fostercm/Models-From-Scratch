#include <hip/hip_runtime.h>
#include <memory_functions.h>
#include <iostream>

// Safe CUDA malloc function
void* safeCudaMalloc(size_t size, int* err) {
    void* d_ptr = NULL;
    *err = hipMalloc(&d_ptr, size);

    if (*err != hipSuccess) {
        printf("CUDA malloc failed: %s\n", hipGetErrorString((hipError_t)(*err)));
        return NULL;  // Return NULL if allocation fails
    }

    return d_ptr;  // Return valid pointer if successful
}

// Safe CUDA free function
void safeCudaFree(void* d_ptr) {
    if (d_ptr) {
        hipFree(d_ptr);
    }
}

// Safe CUDA memcpy function
void safeCudaMemcpy(void* dst, const void* src, size_t count, enum hipMemcpyKind kind) {
    int err = hipMemcpy(dst, src, count, kind);

    if (err != hipSuccess) {
        printf("CUDA memcpy failed: %s\n", hipGetErrorString((hipError_t)err));
    }
}