#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <memory_functions.h>
#include <iostream>

#define MIN(a, b) (a < b ? a : b)

__global__ void populateDiagonalKernel(float *matrix, const float *diagonal, const int m, const int n) {
    // Get thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Populate diagonal (only within the valid range)
    if (tid < MIN(m, n)) {
        if (diagonal[tid] > 1e-15) {
            matrix[tid * n + tid] = 1.0 / diagonal[tid];
        }
    }
}

__global__ void transposeMatrixKernel(const float *matrix, float *transposed, const int m, const int n) {
    // Get thread ID
    int tid = blockIdx.x * blockDim.x + threadIdx.x;

    // Transpose matrix (only within the valid range)
    if (tid < m * n) {
        int i = tid / n;
        int j = tid % n;
        transposed[j * m + i] = matrix[i * n + j];
    }
}

void launchPopulateDiagonalKernel(float *matrix, const float *diagonal, const int m, const int n) {
    // Set up grid and block sizes
    int blockSize = 256;
    int gridSize = (n + blockSize - 1) / blockSize;

    // Launch kernel
    populateDiagonalKernel<<<gridSize, blockSize>>>(matrix, diagonal, m, n);
}

void launchTransposeMatrixKernel(const float *matrix, float *transposed, const int m, const int n) {
    // Set up grid and block sizes
    int blockSize = 256;
    int gridSize = (m * n + blockSize - 1) / blockSize;

    // Launch kernel
    transposeMatrixKernel<<<gridSize, blockSize>>>(matrix, transposed, m, n);
}

// Safe CUDA malloc function
void* safeCudaMalloc(size_t size, int* err) {
    void* d_ptr = NULL;
    *err = hipMalloc(&d_ptr, size);

    if (*err != hipSuccess) {
        printf("CUDA malloc failed: %s\n", hipGetErrorString((hipError_t)(*err)));
        return NULL;  // Return NULL if allocation fails
    }

    return d_ptr;  // Return valid pointer if successful
}

// Safe CUDA free function
void safeCudaFree(void* d_ptr) {
    if (d_ptr) {
        hipFree(d_ptr);
    }
}

// Safe CUDA memcpy function
void safeCudaMemcpy(void* dst, const void* src, size_t count, enum hipMemcpyKind kind) {
    int err = hipMemcpy(dst, src, count, kind);

    if (err != hipSuccess) {
        printf("CUDA memcpy failed: %s\n", hipGetErrorString((hipError_t)err));
    }
}