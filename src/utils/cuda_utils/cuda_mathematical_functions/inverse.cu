/**
 * @file inverse.cu
 * @brief CUDA implementation of inverse and pseudoinverse computation using LU decomposition and SVD respectively.
 *
 * This file provides a CUDA implementation of the inverse and pseudoinverse of a matrix.
 * It uses LU decomposition and Singular Value Decomposition (SVD) respectively. 
 * It leverages cuBLAS and cuSolver libraries to perform matrix operations efficiently on the GPU. 
 * If the matrix is singular, the pseudoinverse is computed instead of the inverse.
 * Intermediate results are stored and managed on the GPU, and memory is freed after each step to avoid memory leaks.
 */

#include "inverse.h"
#include "../cuda_memory_functions/memory_functions.h"
#include "../cuda_matrix_functions/matrix_functions.h"
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>

#define MIN(a, b) (((a) < (b)) ? (a) : (b))

void computeInverse(float *d_A, float *d_A_inv, int n, hipblasHandle_t cublasHandle) {
    // Initialize the cuSolver library
    hipsolverHandle_t cusolverHandle;
    hipsolverDnCreate(&cusolverHandle);

    // Initialize error variable
    int err = 0;

    // Initialize the workspace
    int *d_info, *d_pivot;
    d_info = (int*) safeCudaMalloc(sizeof(int), &err);
    d_pivot = (int*) safeCudaMalloc(n * sizeof(int), &err);

    // Initialize the workspace size
    int lwork = 0;
    int info = 0;
    float *d_work;
    hipsolverDnSgetrf_bufferSize(cusolverHandle, n, n, d_A_inv, n, &lwork);
    d_work = (float*) safeCudaMalloc(lwork * sizeof(float), &err);

    // Perform LU decomposition
    hipsolverDnSgetrf(cusolverHandle, n, n, d_A_inv, n, d_work, d_pivot, d_info);
    safeCudaMemcpy(&info, d_info, sizeof(int), hipMemcpyDeviceToHost);

    // Check if matrix is singular
    if (info != 0) {
        // Compute pseudoinverse
        computePseudoinverse(d_A, d_A_inv, n, n, cublasHandle, cusolverHandle);
    }
    else {
        // Compute identity matrix
        float *d_identity;
        d_identity = (float*) safeCudaMalloc(n * n * sizeof(float), &err);
        launchIdentityMatrixKernel(d_identity, n, n);

        // Solve the system
        hipsolverDnSgetrs(cusolverHandle, HIPBLAS_OP_N, n, n, d_A_inv, n, d_pivot, d_identity, n, d_info);

        // Free memory
        safeCudaFree(d_identity);
    }

    // Free memory
    safeCudaFree(d_info);
    safeCudaFree(d_pivot);
    safeCudaFree(d_work);

    // Destroy the cuSolver handle
    hipsolverDnDestroy(cusolverHandle);
}

/**
 * @brief Computes the pseudoinverse of a matrix using CUDA.
 *
 * This function computes the pseudoinverse of a matrix `A` using Singular Value Decomposition (SVD) 
 * and the CUDA cuBLAS and cuSolver libraries. The matrix is transposed to ensure the operations are performed 
 * in a column-major format. The SVD is computed to find the singular values and vectors, and the pseudoinverse 
 * is obtained by inverting the non-zero singular values. The result is stored in `d_A_inv`.
 *
 * @param[in] d_A Pointer to the input matrix `A` stored on the GPU with size `m * n`.
 * @param[out] d_A_inv Pointer to the resulting pseudoinverse matrix, also stored on the GPU with size `n * m`.
 * @param[in] m The number of rows in matrix `A`.
 * @param[in] n The number of columns in matrix `A`.
 * @param[in] cublasH Handle to the cuBLAS library used for matrix operations.
 *
 * @note This function uses cuBLAS for matrix operations and cuSolver for computing the SVD of the matrix.
 *       Memory is allocated and freed for intermediate results throughout the process.
 */
 void computePseudoinverse(float *d_A, float *d_A_inv, const int m, const int n, hipblasHandle_t cublasH, hipsolverHandle_t cusolverH) {
    // Initialize alpha and beta
    const float alpha = 1.0f;
    const float beta = 0.0f;

    // Initialize error pointer for safe memory allocation
    int err = 0;

    // Allocate memory for the SVD results
    float *d_U_T, *d_S, *d_V;
    d_U_T = (float*) safeCudaMalloc(m * m * sizeof(float), &err);
    d_S = (float*) safeCudaMalloc(MIN(m, n) * sizeof(float), &err);
    d_V = (float*) safeCudaMalloc(n * n * sizeof(float), &err);

    // Allocate memory for SVD info
    int *devInfo;
    int lwork = 0;
    float *d_work;
    devInfo = (int*) safeCudaMalloc(sizeof(int), &err);
    hipsolverDnSgesvd_bufferSize(cusolverH, m, n, &lwork);
    d_work = (float*) safeCudaMalloc(lwork * sizeof(float), &err);

    // Transpose A to make it column-major
    float *d_A_T;
    d_A_T = (float*) safeCudaMalloc(m * n * sizeof(float), &err);
    hipblasSgeam(
        cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, 
        m, n, 
        &alpha, 
        d_A, n, 
        &beta, 
        d_A, m, 
        d_A_T, m
        );

    // Compute SVD
    hipsolverDnSgesvd(cusolverH, 'A', 'A', m, n, d_A_T, m, d_S, d_U_T, m, d_V, n, d_work, lwork, NULL, devInfo);

    // Free SVD info
    safeCudaFree(d_work);
    safeCudaFree(devInfo);

    // Initialize S_inv
    float *d_S_inv;
    d_S_inv = (float*) safeCudaMalloc(m * n * sizeof(float), &err);
    hipMemset(d_S_inv, 0, m * n * sizeof(float));

    // Populate the diagonal of S_inv
    launchPopulateDiagonalKernel(d_S_inv, d_S, m, n);

    // Free S
    safeCudaFree(d_S);

    // Compute VS_inv = V * S_inv and free V & S_inv
    float *d_VS_inv;
    d_VS_inv = (float*) safeCudaMalloc(n * m * sizeof(float), &err);
    hipblasSgemm(
        cublasH, HIPBLAS_OP_T, HIPBLAS_OP_N, 
        n, m, n, 
        &alpha, 
        d_V, n, 
        d_S_inv, n, 
        &beta, 
        d_VS_inv, n
        );
    safeCudaFree(d_V);
    safeCudaFree(d_S_inv);

    // Compute A_inv = VS_inv * U_T and free U_T & VS_inv
    hipblasSgemm(
        cublasH, HIPBLAS_OP_N, HIPBLAS_OP_T, 
        n, m, m, 
        &alpha, 
        d_VS_inv, n, 
        d_U_T, m, 
        &beta, 
        d_A_inv, n
        );
    safeCudaFree(d_U_T);
    safeCudaFree(d_VS_inv);
}