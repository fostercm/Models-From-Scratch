#include <hip/hip_runtime.h>
#include <hipblas.h>
#include "cuda_memory_functions/memory_functions.h"
#include "cuda_mathematical_functions/activation.h"
#include "cuda_loss_functions/loss_functions.h"
#include "logistic_regression.h"

void fit(const float *X, const float *Y, float *Beta, const int n_samples, const int n_input_features, const int n_classes, const int max_iters, float lr, const float tol) {
    // Initialize cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);
    
    // Initialize error variable
    int err = 0;

    // Initialize device variables
    float *d_X, *d_Y, *d_Beta, *d_Gradient, *d_Prediction;
    d_X = (float*) safeCudaMalloc(n_samples * n_input_features * sizeof(float), &err);
    d_Y = (float*) safeCudaMalloc(n_samples * n_classes * sizeof(float), &err);
    d_Beta = (float*) safeCudaMalloc(n_input_features * n_classes * sizeof(float), &err);
    d_Gradient = (float*) safeCudaMalloc(n_input_features * n_classes * sizeof(float), &err);
    d_Prediction = (float*) safeCudaMalloc(n_samples * n_classes * sizeof(float), &err);
    float loss = 0.0f;
    float prev_loss = 0.0f;

    // Transfer X, Y, and Beta to device
    safeCudaMemcpy(d_X, X, n_samples * n_input_features * sizeof(float), hipMemcpyHostToDevice);
    safeCudaMemcpy(d_Y, Y, n_samples * n_classes * sizeof(float), hipMemcpyHostToDevice);
    safeCudaMemcpy(d_Beta, Beta, n_input_features * n_classes * sizeof(float), hipMemcpyHostToDevice);

    // Initialize alpha and beta
    float alpha = 1.0f;
    float beta = 0.0f;
    float gamma = -1.0f;

    // Modify learning rate
    lr = -lr / n_samples;

    for (int i=0; i < max_iters; i++) {
        // Predict
        _predict(d_X, d_Beta, d_Prediction, n_samples, n_input_features, n_classes, handle);

        // Check for convergence
        if (i % 1000 == 0) {
            // Compute cost
            loss = cost(d_Prediction, d_Y, n_samples, n_classes);

            // If loss is less than tolerance, break
            if (i > 0 && loss / prev_loss < tol) {
                break;
            }
            prev_loss = loss;
        }

        // Calculate the difference between the prediction and the true values (in place)
        hipblasSaxpy(handle, n_samples*n_classes, &gamma, d_Y, 1, d_Prediction, 1);

        // Multiply the transpose of the input matrix by the difference (compute the gradient)
        hipblasSgemm(
            handle, HIPBLAS_OP_N, HIPBLAS_OP_T,
            n_classes, n_input_features, n_samples,
            &alpha,
            d_Prediction, n_classes, // row major
            d_X, n_samples, // row major
            &beta,
            d_Gradient, n_classes // row major
        );

        // Update Beta
        hipblasSaxpy(handle, n_input_features*n_classes, &lr, d_Gradient, 1, d_Beta, 1);
    }

    // Transfer Beta to host
    safeCudaMemcpy(Beta, d_Beta, n_input_features * n_classes * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    safeCudaFree(d_X);
    safeCudaFree(d_Y);
    safeCudaFree(d_Beta);
    safeCudaFree(d_Gradient);
    safeCudaFree(d_Prediction);

    // Destroy cublas handle
    hipblasDestroy(handle);
}

void predict(const float *X, const float *Beta, float *Prediction, const int n_samples, const int n_input_features, const int n_classes) {
    // Initialize cublas handle
    hipblasHandle_t handle;
    hipblasCreate(&handle);

    // Initialize error variable
    int err = 0;

    // Initialize device variables
    float *d_X, *d_Beta, *d_Prediction;
    d_X = (float*) safeCudaMalloc(n_samples * n_input_features * sizeof(float), &err);
    d_Beta = (float*) safeCudaMalloc(n_input_features * n_classes * sizeof(float), &err);
    d_Prediction = (float*) safeCudaMalloc(n_samples * n_classes * sizeof(float), &err);

    // Transfer X and Beta to device
    safeCudaMemcpy(d_X, X, n_samples * n_input_features * sizeof(float), hipMemcpyHostToDevice);
    safeCudaMemcpy(d_Beta, Beta, n_input_features * n_classes * sizeof(float), hipMemcpyHostToDevice);

    // Predict
    _predict(d_X, d_Beta, d_Prediction, n_samples, n_input_features, n_classes, handle);

    // Transfer prediction to host
    safeCudaMemcpy(Prediction, d_Prediction, n_samples * n_classes * sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    safeCudaFree(d_X);
    safeCudaFree(d_Beta);
    safeCudaFree(d_Prediction);

    // Destroy cublas handle
    hipblasDestroy(handle);
}

void _predict(const float *d_X, const float *d_Beta, float *d_Prediction, const int n_samples, const int n_input_features, const int n_classes, hipblasHandle_t handle) {
    // Initialize alpha and beta
    float alpha = 1.0f;
    float beta = 0.0f;

    // Compute X * Beta
    hipblasSgemm(
        handle, HIPBLAS_OP_N, HIPBLAS_OP_N,
        n_classes, n_samples, n_input_features,
        &alpha,
        d_Beta, n_classes, // row major
        d_X, n_input_features, // row major
        &beta,
        d_Prediction, n_classes // row major
    );

    // Apply activation
    if (n_classes == 1) {
        // Sigmoid
        sigmoid(d_Prediction, n_samples);
    } else {
        // Softmax
        softmax(d_Prediction, n_samples, n_classes);
    }
}

float cost(const float *Y_pred, const float *Y, const int n_samples, const int n_classes) {
    // Initialize error variable
    int err = 0;

    // Initialize cost
    float cost = 0.0f;

    // Initialize device variables
    float *d_Y_pred, *d_Y, *d_cost;
    d_Y_pred = (float*) safeCudaMalloc(n_samples * n_classes * sizeof(float), &err);
    d_Y = (float*) safeCudaMalloc(n_samples * n_classes * sizeof(float), &err);
    d_cost = (float*) safeCudaMalloc(sizeof(float), &err);
    safeCudaMemcpy(d_cost, &cost, sizeof(float), hipMemcpyHostToDevice);

    // Transfer Y_pred and Y to device
    safeCudaMemcpy(d_Y_pred, Y_pred, n_samples * n_classes * sizeof(float), hipMemcpyHostToDevice);
    safeCudaMemcpy(d_Y, Y, n_samples * n_classes * sizeof(float), hipMemcpyHostToDevice);

    // Compute cost
    crossEntropy(d_Y_pred, d_Y, d_cost, n_samples, n_classes);

    // Transfer cost to host
    safeCudaMemcpy(&cost, d_cost, sizeof(float), hipMemcpyDeviceToHost);

    // Free device memory
    safeCudaFree(d_Y_pred);
    safeCudaFree(d_Y);
    safeCudaFree(d_cost);

    return cost;
}